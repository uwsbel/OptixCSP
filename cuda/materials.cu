#include "hip/hip_runtime.h"
#include <optix.h>
#include <hip/hip_vector_types.h>
#include "Soltrace.h"

// Launch parameters for soltrace
extern "C" {
    __constant__ soltrace::LaunchParams params;
}

static __device__ __inline__ soltrace::PerRayData getPayload()
{
    soltrace::PerRayData prd;
    prd.ray_path_index = optixGetPayload_0();
    prd.depth = optixGetPayload_1();
    return prd;
}

static __device__ __inline__ void setPayload(const soltrace::PerRayData& prd)
{
    optixSetPayload_0(prd.ray_path_index);
    optixSetPayload_1(prd.depth);
}

extern "C" __global__ void __closesthit__mirror()
{
    //const soltrace::HitGroupData* sbt_data = reinterpret_cast<soltrace::HitGroupData*>(optixGetSbtDataPointer());
    //const MaterialData::Mirror& mirror = sbt_data->material_data.mirror;

    // Fetch the normal vector from the hit attributes passed by OptiX
    float3 object_normal = make_float3( __uint_as_float( optixGetAttribute_0() ), __uint_as_float( optixGetAttribute_1() ),
                                        __uint_as_float( optixGetAttribute_2() ) );
    // Transform the object-space normal to world space using OptiX built-in function
    float3 world_normal  = normalize( optixTransformNormalFromObjectToWorldSpace( object_normal ) );
    // Compute the facing normal, which handles the direction of the normal based on the incoming ray direction
    float3 ffnormal      = faceforward( world_normal, -optixGetWorldRayDirection(), world_normal );

    // Get the incoming ray's origin, direction, and max t (intersection distance)
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();
    const float  ray_t    = optixGetRayTmax();

    // Compute the hit point of the ray using its origin and direction, scaled by the intersection distance (ray_t)
    const float3 hit_point = ray_orig + ray_t * ray_dir;

    soltrace::PerRayData prd = getPayload();
    const int new_depth = prd.depth + 1;    // Increment the ray depth for recursive tracing

    // Calculate ideal reflection direction using OptiX's built-in reflect function
    float3 reflected_dir = reflect(ray_dir, ffnormal);

    // TODO: Add some noise here

    // Check if the maximum recursion depth has not been reached
    if (new_depth < params.max_depth) {
        // Store the hit point in the hit point buffer (used for visualization or further calculations)
        params.hit_point_buffer[params.max_depth * prd.ray_path_index + new_depth] = make_float4(1.0f, hit_point);
        // Store the reflected direction in its buffer (used for visualization or further calculations)
        /*
        params.reflected_dir_buffer[params.max_depth * prd.ray_path_index + new_depth] = make_float4(1.0f, reflected_dir);
        */

        // Trace the reflected ray
        prd.depth = new_depth;
        optixTrace(
            params.handle,          // The handle to the acceleration structure
            hit_point,              // The starting point of the reflected ray
            reflected_dir,          // The direction of the reflected ray
            0.01f,                  // A small offset to avoid self-intersection (shadow acne)
            1e16f,                  // Maximum distance the ray can travel
            0.0f,                   // Ray time (used for time-dependent effects)
            OptixVisibilityMask(1), // Visibility mask (defines what the ray can interact with)
            OPTIX_RAY_FLAG_NONE,    // Ray flags (no special flags for now)
            soltrace::RAY_TYPE_RADIANCE,  // Use the radiance ray type
            soltrace::RAY_TYPE_COUNT,     // Total number of ray types
            soltrace::RAY_TYPE_RADIANCE,  // The ray type's offset into the SBT
            reinterpret_cast<unsigned int&>(prd.ray_path_index), // Pass the ray path index
            reinterpret_cast<unsigned int&>(prd.depth)           // Pass the updated depth
        );
    }

    setPayload(prd);
}

extern "C" __global__ void __closesthit__receiver()
{
    // Retrieve the hit group data and access the parallelogram geometry
    const soltrace::HitGroupData* sbt_data = reinterpret_cast<soltrace::HitGroupData*>(optixGetSbtDataPointer());
    const GeometryData::Parallelogram& parallelogram = sbt_data->geometry_data.getParallelogram();

    /*
    float3 object_normal = make_float3( __uint_as_float( optixGetAttribute_0() ), __uint_as_float( optixGetAttribute_1() ),
                                        __uint_as_float( optixGetAttribute_2() ) );
    float3 world_normal  = normalize( optixTransformNormalFromObjectToWorldSpace( object_normal ) );
    float3 ffnormal      = faceforward( world_normal, -optixGetWorldRayDirection(), world_normal );
    */

    // Incident ray properties (origin, direction, and max t distance)
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();
    const float  ray_t    = optixGetRayTmax();

    // Compute the normal of the receiver and dot with ray direction to determine which side was hit
    // TODO: this normal is hard coded based on how the geometry was defined, need to make more robust
    const float3 receiver_normal = cross(parallelogram.v2, parallelogram.v1);
    const float dot_product = dot(ray_dir, receiver_normal);

    float3 hit_point = ray_orig + ray_t * ray_dir;

    soltrace::PerRayData prd = getPayload();
    const int new_depth = prd.depth + 1;

    // Check if the ray hits the receiver surface (dot product negative means ray is hitting the front face)
    if (dot_product < 0.0f) {
        if (new_depth < params.max_depth) {
            params.hit_point_buffer[params.max_depth * prd.ray_path_index + new_depth] = make_float4(2.0f, hit_point);
            prd.depth = new_depth;
        }
    }

    setPayload(prd);
}

extern "C" __global__ void __closesthit__receiver__cylinder__y()
{
    // Retrieve the hit group data and access the parallelogram geometry
    const soltrace::HitGroupData* sbt_data = reinterpret_cast<soltrace::HitGroupData*>(optixGetSbtDataPointer());
    const GeometryData::Cylinder_Y& cyl = sbt_data->geometry_data.getCylinder_Y();

    /*
    float3 object_normal = make_float3( __uint_as_float( optixGetAttribute_0() ), __uint_as_float( optixGetAttribute_1() ),
                                        __uint_as_float( optixGetAttribute_2() ) );
    float3 world_normal  = normalize( optixTransformNormalFromObjectToWorldSpace( object_normal ) );
    float3 ffnormal      = faceforward( world_normal, -optixGetWorldRayDirection(), world_normal );
    */

    // Incident ray properties (origin, direction, and max t distance)
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir = optixGetWorldRayDirection();
    const float  ray_t = optixGetRayTmax();

    // Compute the normal of the receiver and dot with ray direction to determine which side was hit
    // TODO: this normal is hard coded based on how the geometry was defined, need to make more robust
    //const float3 receiver_normal = cyl.base_x;
    //const float dot_product = dot(ray_dir, receiver_normal);

    float3 hit_point = ray_orig + ray_t * ray_dir;

    soltrace::PerRayData prd = getPayload();
    const int new_depth = prd.depth + 1;

    // Check if the ray hits the receiver surface (dot product negative means ray is hitting the front face)
    //if (dot_product < 0.0f) {
        if (new_depth < params.max_depth) {
            params.hit_point_buffer[params.max_depth * prd.ray_path_index + new_depth] = make_float4(2.0f, hit_point);
            prd.depth = new_depth;
        }
    //}

    setPayload(prd);
}


// Closest-hit for a parabolic mirror surface.
// This function is nearly identical to __closesthit__mirror(), but is intended to be
// used when the surface is a parabolic mirror. The intersection shader (e.g. 
// __intersection__rectangle_parabolic) reports a normal that already accounts for the curvature.
extern "C" __global__ void __closesthit__mirror__parabolic()
{
    // Optionally, you can access material data if needed:
    // const soltrace::HitGroupData* sbt_data = reinterpret_cast<soltrace::HitGroupData*>( optixGetSbtDataPointer() );
    // const MaterialData::Mirror& mirror = sbt_data->material_data.mirror;

    // Retrieve the hit normal from the attributes.
    // The intersection shader for the parabolic surface reported the normal (using float3_as_args)
    // in the hit attributes. In many cases this normal is already in world space.
    float3 object_normal = make_float3(__uint_as_float(optixGetAttribute_0()),
        __uint_as_float(optixGetAttribute_1()),
        __uint_as_float(optixGetAttribute_2()));
    // If the normal reported by the intersection shader is in object space you can transform it;
    // if it is already in world space, this call may be omitted.
    float3 world_normal = normalize(optixTransformNormalFromObjectToWorldSpace(object_normal));

    // Ensure that the normal is facing the incoming ray.
    float3 ffnormal = faceforward(world_normal, -optixGetWorldRayDirection(), world_normal);

    // Retrieve ray data.
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir = optixGetWorldRayDirection();
    const float  ray_t = optixGetRayTmax();

    // Compute the hit point.
    const float3 hit_point = ray_orig + ray_t * ray_dir;

    // Retrieve per�ray payload.
    soltrace::PerRayData prd = getPayload();
    const int new_depth = prd.depth + 1; // Increase recursion depth.

    // Compute the reflected ray direction.
    float3 reflected_dir = reflect(ray_dir, ffnormal);

    // (Optional: Add noise to reflected_dir if desired.)

    // If the new depth is below the maximum, trace the reflected ray.
    if (new_depth < params.max_depth) {
        // Save the hit point (for visualization or further processing).
        params.hit_point_buffer[params.max_depth * prd.ray_path_index + new_depth] = make_float4(1.0f, hit_point);

        prd.depth = new_depth;
        optixTrace(
            params.handle,          // Acceleration structure handle.
            hit_point,              // Ray origin.
            reflected_dir,          // Ray direction.
            0.01f,                  // Minimum t to avoid self-intersection.
            1e16f,                  // Maximum t.
            0.0f,                   // Ray time.
            OptixVisibilityMask(1), // Visibility mask.
            OPTIX_RAY_FLAG_NONE,    // Ray flags.
            soltrace::RAY_TYPE_RADIANCE,  // Ray type.
            soltrace::RAY_TYPE_COUNT,     // Number of ray types.
            soltrace::RAY_TYPE_RADIANCE,  // SBT offset for this ray type.
            reinterpret_cast<unsigned int&>(prd.ray_path_index), // Ray path index.
            reinterpret_cast<unsigned int&>(prd.depth)           // Current recursion depth.
        );
    }

    // Store the updated payload.
    setPayload(prd);
}




extern "C" __global__ void __miss__ms()
{
    // No action is taken here.
    // This function simply acts as a terminator for rays that miss all geometry.
    
    /*
    soltrace::PerRayData prd = getPayload();
    const int new_depth = prd.depth + 1;

    if (new_depth < params.max_depth) {
        params.hit_point_buffer[params.max_depth * ray_path_index + new_depth] = make_float4(4.0f);
    }
    */

    // Set the payload values to 0, indicating that the ray missed all geometry.
    optixSetPayload_0(0);  // Default value
    optixSetPayload_1(0);
}