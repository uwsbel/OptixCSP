#include "hip/hip_runtime.h"
#include <optix.h>
#include <cuda/helpers.h>
#include "Soltrace.h"
#include <stdio.h>

extern "C" __global__ void __intersection__parallelogram()
{
    // Load shader binding table (SBT) and access data specific to this hit group
    const soltrace::HitGroupData* sbt_data = reinterpret_cast<soltrace::HitGroupData*>(optixGetSbtDataPointer());
    const GeometryData::Parallelogram& parallelogram = sbt_data->geometry_data.getParallelogram();

    // Get ray information: origin, direction, and min/max distances over which ray should be tested
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();
    const float  ray_tmin = optixGetRayTmin(), ray_tmax = optixGetRayTmax();

    // Compute ray intersection point
    float3 n  = make_float3( parallelogram.plane );
    float  dt = dot( ray_dir, n );
    // Compute distance t (point of intersection) along ray direction from ray origin
    float  t  = ( parallelogram.plane.w - dot( n, ray_orig ) ) / dt;

    // Verify intersection distance and Report ray intersection point
    if( t > ray_tmin && t < ray_tmax )
    {
        float3 p  = ray_orig + ray_dir * t;
        float3 vi = p - parallelogram.anchor;
        float  a1 = dot( parallelogram.v1, vi );
        if( a1 >= 0 && a1 <= 1 )
        {
            float a2 = dot( parallelogram.v2, vi );
            if( a2 >= 0 && a2 <= 1 )
            {
                // float as uint? is this a boolean? 
                optixReportIntersection( t, 0, float3_as_args( n ), __float_as_uint( a1 ), __float_as_uint( a2 ) );
            }
        }
    }
}

extern "C" __global__ void __intersection__cylinder_y()
{
    // Load shader binding table (SBT) and access data specific to this hit group
    const soltrace::HitGroupData* sbt_data = reinterpret_cast<soltrace::HitGroupData*>(optixGetSbtDataPointer());
    const GeometryData::Cylinder_Y& cyl = sbt_data->geometry_data.getCylinder_Y();

    // Get ray information: origin, direction, and min/max distances over which ray should be tested
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir = normalize(optixGetWorldRayDirection());
    const float  ray_tmin = optixGetRayTmin();
    const float  ray_tmax = optixGetRayTmax();

    // Transform ray to the cylinder's local coordinate system
    float3 local_ray_orig = ray_orig - cyl.center;
    float3 local_ray_dir = ray_dir;

	// TODO: check how to optimize this, there should be a way in optix to rotate coordinates 
    float3 local_x = cyl.base_x;
    float3 local_z = cyl.base_z;
    float3 local_y = cross(local_z, local_x);

    local_ray_orig = make_float3(
        dot(local_ray_orig, local_x),
        dot(local_ray_orig, local_y),
        dot(local_ray_orig, local_z)
    );
    local_ray_dir = make_float3(
        dot(local_ray_dir, local_x),
        dot(local_ray_dir, local_y),
        dot(local_ray_dir, local_z)
    );

	// solve quadratic equation for intersection
    float A = local_ray_dir.x * local_ray_dir.x + local_ray_dir.z * local_ray_dir.z;
    float B = 2.0f * (local_ray_orig.x * local_ray_dir.x + local_ray_orig.z * local_ray_dir.z);
    float C = local_ray_orig.x * local_ray_orig.x + local_ray_orig.z * local_ray_orig.z - cyl.radius * cyl.radius;

    float determinant = B * B - 4.0f * A * C;

    if (determinant < 0.0f)
    {
        // No intersection
        return;
    }

    // Compute intersection distances
    float t1 = (-B - sqrtf(determinant)) / (2.0f * A);
    float t2 = (-B + sqrtf(determinant)) / (2.0f * A);

    float t = t1 > 0.0f ? t1 : t2; // Use the closer valid intersection
    if (t < ray_tmin || t > ray_tmax)
    {
        // Intersection is out of bounds
        return;
    }

    // Compute intersection point in local space
    float3 local_hit_point = local_ray_orig + t * local_ray_dir;

    // Check if the hit point is within the cylinder's height bounds
    if (fabsf(local_hit_point.y) > cyl.half_height)
    {
        // If t1 is invalid, try t2
        t = t2;
        local_hit_point = local_ray_orig + t * local_ray_dir;
        if (t < ray_tmin || t > ray_tmax || fabsf(local_hit_point.y) > cyl.half_height)
        {
            return; // Both intersections are out of bounds
        }
    }

    // Compute normal in local coordinates
    float3 local_normal = normalize(make_float3(local_hit_point.x, 0.0f, local_hit_point.z));

    // Transform normal back to world coordinates
    float3 world_normal = local_normal.x * local_x + local_normal.y * local_y + local_normal.z * local_z;

    // Compute the hit point in world space
    float3 world_hit_point = ray_orig + t * ray_dir;

    // Report intersection to OptiX
    optixReportIntersection(
        t,
        0,
        float3_as_args(world_normal),
        __float_as_uint(world_hit_point.x),
        __float_as_uint(world_hit_point.y)
    );
}

// ray cylinder intersection with top and bottom caps 
// it can also be modeled as cylinder with two disks. 
extern "C" __global__ void __intersection__cylinder_y_capped()
{
    // Load shader binding table (SBT) and access data specific to this hit group
    const soltrace::HitGroupData* sbt_data = reinterpret_cast<soltrace::HitGroupData*>(optixGetSbtDataPointer());
    const GeometryData::Cylinder_Y& cyl = sbt_data->geometry_data.getCylinder_Y();

    // Get ray information: origin, direction, and min/max distances over which ray should be tested
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir = normalize(optixGetWorldRayDirection());
    const float  ray_tmin = optixGetRayTmin();
    const float  ray_tmax = optixGetRayTmax();

    // Transform ray to the cylinder's local coordinate system
    float3 local_ray_orig = ray_orig - cyl.center;
    float3 local_ray_dir = ray_dir;

    // Transform using the cylinder's local basis
    float3 local_x = cyl.base_x;
    float3 local_z = cyl.base_z;
    float3 local_y = cross(local_z, local_x);

    local_ray_orig = make_float3(
        dot(local_ray_orig, local_x),
        dot(local_ray_orig, local_y),
        dot(local_ray_orig, local_z)
    );
    local_ray_dir = make_float3(
        dot(local_ray_dir, local_x),
        dot(local_ray_dir, local_y),
        dot(local_ray_dir, local_z)
    );

    // Solve quadratic equation for intersection with curved surface
    float A = local_ray_dir.x * local_ray_dir.x + local_ray_dir.z * local_ray_dir.z;
    float B = 2.0f * (local_ray_orig.x * local_ray_dir.x + local_ray_orig.z * local_ray_dir.z);
    float C = local_ray_orig.x * local_ray_orig.x + local_ray_orig.z * local_ray_orig.z - cyl.radius * cyl.radius;

    float determinant = B * B - 4.0f * A * C;

    float t_curved = ray_tmax + 1.0f; // Initialize to invalid
    if (determinant >= 0.0f)
    {
        // Compute intersection distances
        float t1 = (-B - sqrtf(determinant)) / (2.0f * A);
        float t2 = (-B + sqrtf(determinant)) / (2.0f * A);

        // Select the closest valid intersection within bounds
        if (t1 > ray_tmin && t1 < ray_tmax && fabsf(local_ray_orig.y + t1 * local_ray_dir.y) <= cyl.half_height)
        {
            t_curved = t1;
        }
        else if (t2 > ray_tmin && t2 < ray_tmax && fabsf(local_ray_orig.y + t2 * local_ray_dir.y) <= cyl.half_height)
        {
            t_curved = t2;
        }
    }

    // Check intersection with top and bottom caps
    float t_caps = ray_tmax + 1.0f;
    {
        // Bottom cap: y = -half_height
        if (fabsf(local_ray_dir.y) > 1e-6f) // Avoid division by zero
        {
            float t = (-cyl.half_height - local_ray_orig.y) / local_ray_dir.y;
            float2 hit_point = make_float2(local_ray_orig.x + t * local_ray_dir.x,
                local_ray_orig.z + t * local_ray_dir.z);
            if (t > ray_tmin && t < ray_tmax && dot(hit_point, hit_point) <= cyl.radius * cyl.radius)
            {
                t_caps = t;
            }
        }

        // Top cap: y = +half_height
        if (fabsf(local_ray_dir.y) > 1e-6f)
        {
            float t = (cyl.half_height - local_ray_orig.y) / local_ray_dir.y;
            float2 hit_point = make_float2(local_ray_orig.x + t * local_ray_dir.x,
                local_ray_orig.z + t * local_ray_dir.z);
            if (t > ray_tmin && t < ray_tmax && dot(hit_point, hit_point) <= cyl.radius * cyl.radius)
            {
                t_caps = fminf(t_caps, t);
            }
        }
    }

    // Use the closest valid intersection
    float t = fminf(t_curved, t_caps);
    if (t >= ray_tmax || t <= ray_tmin)
    {
        return; // No valid intersection
    }

    // Compute intersection point and normal
    float3 local_hit_point = local_ray_orig + t * local_ray_dir;
    float3 local_normal;

    if (t == t_curved)
    {
        // Hit on the curved surface
        local_normal = normalize(make_float3(local_hit_point.x, 0.0f, local_hit_point.z));
    }
    else
    {
        // Hit on one of the caps
        local_normal = make_float3(0.0f, signbit(local_hit_point.y) ? -1.0f : 1.0f, 0.0f);
    }

    // Transform normal back to world coordinates
    float3 world_normal = local_normal.x * local_x + local_normal.y * local_y + local_normal.z * local_z;

    // Compute world-space hit point
    float3 world_hit_point = ray_orig + t * ray_dir;

    // Report intersection to OptiX
    optixReportIntersection(
        t,
        0, // User-defined instance ID or custom data
        float3_as_args(world_normal),
        __float_as_uint(world_hit_point.x),
        __float_as_uint(world_hit_point.y)
    );
}


#include <optix.h>
#include <cuda/helpers.h>
#include "Soltrace.h"

extern "C" __global__ void __intersection__rectangle_parabolic()
{
    // Load shader binding table (SBT) data and retrieve the parabolic rectangle.
    const soltrace::HitGroupData* sbt_data = reinterpret_cast<soltrace::HitGroupData*>(optixGetSbtDataPointer());
    const GeometryData::Rectangle_Parabolic& rect = sbt_data->geometry_data.getRectangleParabolic();

    // Get ray information.
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir = optixGetWorldRayDirection();
    const float  ray_tmin = optixGetRayTmin();
    const float  ray_tmax = optixGetRayTmax();

    float L1 = 1.0f / length(rect.v1);
    float L2 = 1.0f / length(rect.v2);
    // And the unit edge directions are:
    float3 e1 = rect.v1 * L1; // recovers the original direction of edge 1
    float3 e2 = rect.v2 * L2; // recovers the original direction of edge 2
    // The flat (undeformed) rectangle�s normal is:
    float3 n = normalize(cross(e2, e1));

    // Transform ray into local coordinates.
    float3 d = ray_orig - rect.anchor;
    float ox = dot(d, e1);
    float oy = dot(d, e2);
    float oz = dot(d, n);

    float dx = dot(ray_dir, e1);
    float dy = dot(ray_dir, e2);
    float dz = dot(ray_dir, n);

    // Retrieve curvature parameters.
    const float curv_x = rect.curv_x;
    const float curv_y = rect.curv_y;

    float A = (curv_x * 0.5f) * (dx * dx) + (curv_y * 0.5f) * (dy * dy);
    float B = curv_x * (ox * dx) + curv_y * (oy * dy) - dz;
    float C = (curv_x * 0.5f) * (ox * ox) + (curv_y * 0.5f) * (oy * oy) - oz;

    float t = 0.0f;
    const float eps = 1e-12f;
    bool valid = false;

    if (fabsf(A) < eps) {
        // Degenerate (linear) case.
        t = -C / B;
        valid = (t > 0.0f);
    }
    else {
        float discr = B * B - 4.0f * A * C;
        if (discr >= 0.0f) {
            float sqrt_discr = sqrtf(discr);
            float t1 = (-B - sqrt_discr) / (2.0f * A);
            float t2 = (-B + sqrt_discr) / (2.0f * A);
            // Choose the smallest positive t.
            if (t1 > 0.0f && t1 < t2) {
                t = t1;
                valid = true;
            }
            else if (t2 > 0.0f) {
                t = t2;
                valid = true;
            }
        }
    }

    // Discard if no valid t or if t is not within the ray�s bounds.
    if (!valid || t < ray_tmin || t > ray_tmax) {
        return;
    }

    //
    // Compute the local intersection coordinates.
    //
    float x_hit = ox + t * dx;
    float y_hit = oy + t * dy;

    // Check if the hit is within the rectangle�s flat bounds.
    float a1 = x_hit / L1;
    float a2 = y_hit / L2;
    if (a1 < 0.0f || a1 > 1.0f || a2 < 0.0f || a2 > 1.0f) {
        return;
    }

    // The height function is:
    //    f(x,y) = (curv_x/2)*x^2 + (curv_y/2)*y^2
    // so its partial derivatives are:
    //    f_x = curv_x * x    and    f_y = curv_y * y.
    float3 N_local = normalize(make_float3(-curv_x * x_hit,
        -curv_y * y_hit,
        1.0f));
    // Transform the normal back to world coordinates.
    float3 world_normal = normalize(N_local.x * e1 +
        N_local.y * e2 +
        N_local.z * n);

    // Compute the hit point in world space.
    float3 world_hit = ray_orig + t * ray_dir;

    // Report the intersection.
    // Here, the two reported extra attributes are the parametric coordinates (a1, a2),
    // encoded as unsigned integers.
    optixReportIntersection(t, 0,
        float3_as_args(world_normal),
        __float_as_uint(a1),
        __float_as_uint(a2));

    
}
