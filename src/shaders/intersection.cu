#include "hip/hip_runtime.h"
#include <optix.h>
//#include <cuda/helpers.h>
#include "Soltrace.h"
#include <stdio.h>
#include "GeometryDataST.h"

extern "C" {
    __constant__ OptixCSP::LaunchParams params;
}


extern "C" __global__ void __intersection__parallelogram()
{
	int i = optixGetPrimitiveIndex();
    const OptixCSP::GeometryDataST::Parallelogram& parallelogram = params.geometry_data_array[i].getParallelogram();
        
    // Get ray information: origin, direction, and min/max distances over which ray should be tested
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();
    const float  ray_tmin = optixGetRayTmin(), ray_tmax = optixGetRayTmax();

    // Compute ray intersection point
    float3 n  = make_float3( parallelogram.plane );
    float  dt = dot( ray_dir, n );
    // Compute distance t (point of intersection) along ray direction from ray origin
    float  t  = ( parallelogram.plane.w - dot( n, ray_orig ) ) / dt;

    // Verify intersection distance and Report ray intersection point
    if( t > ray_tmin && t < ray_tmax )
    {
        float3 p  = ray_orig + ray_dir * t;
        float3 vi = p - parallelogram.anchor;
        float  a1 = dot( parallelogram.v1, vi );
        if( a1 >= 0 && a1 <= 1 )
        {
            float a2 = dot( parallelogram.v2, vi );
            if( a2 >= 0 && a2 <= 1 )
            {
                optixReportIntersection( t, 
                    0, 
                    __float_as_uint( n.x ), 
                    __float_as_uint( n.y ), 
                    __float_as_uint( n.z ));
            }
        }
    }
}

extern "C" __global__ void __intersection__rectangle_flat()
{

	const OptixCSP::GeometryDataST::Rectangle_Flat& rectangle = params.geometry_data_array[optixGetPrimitiveIndex()].getRectangle_Flat();
        
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir = optixGetWorldRayDirection();
    const float ray_tmin = optixGetRayTmin();
    const float ray_tmax = optixGetRayTmax();

    // Get plane normal and distance
    float3 n = make_float3(rectangle.plane);
    float dt = dot(ray_dir, n);
    
    // Compute distance t (point of intersection) along ray direction from ray origin
    float t = (rectangle.plane.w - dot(n, ray_orig)) / dt;

    // Verify intersection distance
    if (t > ray_tmin && t < ray_tmax)
    {
        // Compute intersection point
        float3 p = ray_orig + ray_dir * t;
        
        // Compute vector from center to intersection point
        float3 v = p - rectangle.center;
        
        // Project onto x and y to get local coordinates
        float x = dot(rectangle.x, v);
        float y = dot(rectangle.y, v);
        
        // Check if point is within rectangle bounds
        if (x >= -rectangle.width/2 && x <= rectangle.width/2 &&
            y >= -rectangle.height/2 && y <= rectangle.height/2)
        {
            optixReportIntersection(t,
                0,
                __float_as_uint(n.x),
                __float_as_uint(n.y),
                __float_as_uint(n.z));
        }
    }
}

extern "C" __global__ void __intersection__cylinder_y()
{
	const OptixCSP::GeometryDataST::Cylinder_Y& cyl = params.geometry_data_array[optixGetPrimitiveIndex()].getCylinder_Y();

    // Get ray information: origin, direction, and min/max distances over which ray should be tested
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir = normalize(optixGetWorldRayDirection());
    const float  ray_tmin = optixGetRayTmin();
    const float  ray_tmax = optixGetRayTmax();

    // Transform ray to the cylinder's local coordinate system
    float3 local_ray_orig = ray_orig - cyl.center;
    float3 local_ray_dir = ray_dir;

	// TODO: check how to optimize this, there should be a way in optix to rotate coordinates 
    float3 local_x = cyl.base_x;
    float3 local_z = cyl.base_z;
    float3 local_y = cross(local_z, local_x);

    local_ray_orig = make_float3(
        dot(local_ray_orig, local_x),
        dot(local_ray_orig, local_y),
        dot(local_ray_orig, local_z)
    );
    local_ray_dir = make_float3(
        dot(local_ray_dir, local_x),
        dot(local_ray_dir, local_y),
        dot(local_ray_dir, local_z)
    );

	// solve quadratic equation for intersection
    float A = local_ray_dir.x * local_ray_dir.x + local_ray_dir.z * local_ray_dir.z;
    float B = 2.0f * (local_ray_orig.x * local_ray_dir.x + local_ray_orig.z * local_ray_dir.z);
    float C = local_ray_orig.x * local_ray_orig.x + local_ray_orig.z * local_ray_orig.z - cyl.radius * cyl.radius;

    float determinant = B * B - 4.0f * A * C;

    if (determinant < 0.0f)
    {
        // No intersection
        return;
    }

    // Compute intersection distances
    float t1 = (-B - sqrtf(determinant)) / (2.0f * A);
    float t2 = (-B + sqrtf(determinant)) / (2.0f * A);

    float t = t1 > 0.0f ? t1 : t2; // Use the closer valid intersection
    if (t < ray_tmin || t > ray_tmax)
    {
        // Intersection is out of bounds
        return;
    }

    // Compute intersection point in local space
    float3 local_hit_point = local_ray_orig + t * local_ray_dir;

    // Check if the hit point is within the cylinder's height bounds
    if (fabsf(local_hit_point.y) > cyl.half_height)
    {
        // If t1 is invalid, try t2
        t = t2;
        local_hit_point = local_ray_orig + t * local_ray_dir;
        if (t < ray_tmin || t > ray_tmax || fabsf(local_hit_point.y) > cyl.half_height)
        {
            return; // Both intersections are out of bounds
        }
    }

    // Compute normal in local coordinates
    float3 local_normal = normalize(make_float3(local_hit_point.x, 0.0f, local_hit_point.z));

    // Transform normal back to world coordinates
    float3 world_normal = local_normal.x * local_x + local_normal.y * local_y + local_normal.z * local_z;

    // Compute the hit point in world space
    float3 world_hit_point = ray_orig + t * ray_dir;

    // Report intersection to OptiX
    optixReportIntersection(t,
        0,
        __float_as_uint(world_normal.x),
        __float_as_uint(world_normal.y),
        __float_as_uint(world_normal.z));
}

// ray cylinder intersection with top and bottom caps 
// it can also be modeled as cylinder with two disks. 
extern "C" __global__ void __intersection__cylinder_y_capped()
{
	const OptixCSP::GeometryDataST::Cylinder_Y& cyl = params.geometry_data_array[optixGetPrimitiveIndex()].getCylinder_Y();

    // Get ray information: origin, direction, and min/max distances over which ray should be tested
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir = normalize(optixGetWorldRayDirection());
    const float  ray_tmin = optixGetRayTmin();
    const float  ray_tmax = optixGetRayTmax();

    // Transform ray to the cylinder's local coordinate system
    float3 local_ray_orig = ray_orig - cyl.center;
    float3 local_ray_dir = ray_dir;

    // Transform using the cylinder's local basis
    float3 local_x = cyl.base_x;
    float3 local_z = cyl.base_z;
    float3 local_y = cross(local_z, local_x);

    local_ray_orig = make_float3(
        dot(local_ray_orig, local_x),
        dot(local_ray_orig, local_y),
        dot(local_ray_orig, local_z)
    );
    local_ray_dir = make_float3(
        dot(local_ray_dir, local_x),
        dot(local_ray_dir, local_y),
        dot(local_ray_dir, local_z)
    );

    // Solve quadratic equation for intersection with curved surface
    float A = local_ray_dir.x * local_ray_dir.x + local_ray_dir.z * local_ray_dir.z;
    float B = 2.0f * (local_ray_orig.x * local_ray_dir.x + local_ray_orig.z * local_ray_dir.z);
    float C = local_ray_orig.x * local_ray_orig.x + local_ray_orig.z * local_ray_orig.z - cyl.radius * cyl.radius;

    float determinant = B * B - 4.0f * A * C;

    float t_curved = ray_tmax + 1.0f; // Initialize to invalid
    if (determinant >= 0.0f)
    {
        // Compute intersection distances
        float t1 = (-B - sqrtf(determinant)) / (2.0f * A);
        float t2 = (-B + sqrtf(determinant)) / (2.0f * A);

        // Select the closest valid intersection within bounds
        if (t1 > ray_tmin && t1 < ray_tmax && fabsf(local_ray_orig.y + t1 * local_ray_dir.y) <= cyl.half_height)
        {
            t_curved = t1;
        }
        else if (t2 > ray_tmin && t2 < ray_tmax && fabsf(local_ray_orig.y + t2 * local_ray_dir.y) <= cyl.half_height)
        {
            t_curved = t2;
        }
    }

    // Check intersection with top and bottom caps
    float t_caps = ray_tmax + 1.0f;
    {
        // Bottom cap: y = -half_height
        if (fabsf(local_ray_dir.y) > 1e-6f) // Avoid division by zero
        {
            float t = (-cyl.half_height - local_ray_orig.y) / local_ray_dir.y;
            float2 hit_point = make_float2(local_ray_orig.x + t * local_ray_dir.x,
                local_ray_orig.z + t * local_ray_dir.z);
            if (t > ray_tmin && t < ray_tmax && dot(hit_point, hit_point) <= cyl.radius * cyl.radius)
            {
                t_caps = t;
            }
        }

        // Top cap: y = +half_height
        if (fabsf(local_ray_dir.y) > 1e-6f)
        {
            float t = (cyl.half_height - local_ray_orig.y) / local_ray_dir.y;
            float2 hit_point = make_float2(local_ray_orig.x + t * local_ray_dir.x,
                local_ray_orig.z + t * local_ray_dir.z);
            if (t > ray_tmin && t < ray_tmax && dot(hit_point, hit_point) <= cyl.radius * cyl.radius)
            {
                t_caps = fminf(t_caps, t);
            }
        }
    }

    // Use the closest valid intersection
    float t = fminf(t_curved, t_caps);
    if (t >= ray_tmax || t <= ray_tmin)
    {
        return; // No valid intersection
    }

    // Compute intersection point and normal
    float3 local_hit_point = local_ray_orig + t * local_ray_dir;
    float3 local_normal;

    if (t == t_curved)
    {
        // Hit on the curved surface
        local_normal = normalize(make_float3(local_hit_point.x, 0.0f, local_hit_point.z));
    }
    else
    {
        // Hit on one of the caps
        local_normal = make_float3(0.0f, signbit(local_hit_point.y) ? -1.0f : 1.0f, 0.0f);
    }

    // Transform normal back to world coordinates
    float3 world_normal = local_normal.x * local_x + local_normal.y * local_y + local_normal.z * local_z;

    // Compute world-space hit point
    float3 world_hit_point = ray_orig + t * ray_dir;

    // Report intersection to OptiX
    optixReportIntersection(
        t,
        0, // User-defined instance ID or custom data
        __float_as_uint(world_normal.x),
        __float_as_uint(world_normal.y),
        __float_as_uint(world_normal.z)
    );
}


// For a parabolic surface rectangle aperture where 
// the base (normal projection) is defined by the center and its two unit edge vectors
// In a local coordinate system (with origin at the anchor) the flat rectangle covers:
//    x in [0, L1]  and  y in [0, L2],
// where L1 and L2 are the lengths of the original edge vectors.
// The parabolic surface is given by:
//    z = (curv_x/2)*x^2 + (curv_y/2)*y^2
// and the ray (in local coordinates) is:
//    (ox,oy,oz) + t*(dx,dy,dz)
// We solve for t such that:
//    oz + t*dz = (curv_x/2)*(ox+t*dx)^2 + (curv_y/2)*(oy+t*dy)^2
// which expands into a quadratic: A*t^2 + B*t + C = 0.
// After finding the valid t, we compute the local hit (x,y) and then check that
//   0 <= x <= L1   and   0 <= y <= L2.
// Finally, we compute the surface normal from the paraboloid derivative
//    f_x = curv_x * x    and    f_y = curv_y * y,
// so that the (unnormalized) local normal is (-f_x, -f_y, 1).
//
// The local hit point is then transformed back to world space for reporting.
extern "C" __global__ void __intersection__rectangle_parabolic()
{
    const OptixCSP::GeometryDataST::Rectangle_Parabolic& rect = params.geometry_data_array[optixGetPrimitiveIndex()].getRectangleParabolic();
    // Get ray information.
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir = optixGetWorldRayDirection();
    const float  ray_tmin = optixGetRayTmin();
    const float  ray_tmax = optixGetRayTmax();

    //
    // Build the local coordinate system.
    //
    // We assume that the rectangle was defined with an anchor at its corner and
    // two edge vectors. The stored rect.v1 and rect.v2 are the reciprocals:
    //     stored_v1 = original_v1 / dot(original_v1, original_v1)
    // Thus, the original edge lengths are:
    // Note rect.v1 had the size of 1/original_v1_length
    float L1 = 1.0f / length(rect.v1);
    float L2 = 1.0f / length(rect.v2);
    // And the unit edge directions are:
    float3 e1 = rect.v1 * L1; // recovers the original direction of edge 1, unit vector
    float3 e2 = rect.v2 * L2; // recovers the original direction of edge 2, unit ve
    // The flat (undeformed) rectangle's normal is:
    float3 n = normalize(cross(e2, e1));

    //
    // Transform ray into local coordinates.
    // The local coordinates (x,y,z) are defined such that:
    //   - The origin is at rect.anchor.
    //   - The x-axis is e1.
    //   - The y-axis is e2.
    //   - The z-axis is n.
    //
    // Compute the rectangle center (shifting from the lower-right corner)
    float3 rect_center = rect.anchor + (L1 / 2.0f) * e1 + (L2 / 2.0f) * e2;

    float3 d = ray_orig - rect_center;
    float ox = dot(d, e1);
    float oy = dot(d, e2);
    float oz = dot(d, n);

    float dx = dot(ray_dir, e1);
    float dy = dot(ray_dir, e2);
    float dz = dot(ray_dir, n);

    // Retrieve curvature parameters.
    const float curv_x = rect.curv_x;
    const float curv_y = rect.curv_y;

    float A = (curv_x * 0.5f) * (dx * dx) + (curv_y * 0.5f) * (dy * dy);
    float B = curv_x * (ox * dx) + curv_y * (oy * dy) - dz;
    float C = (curv_x * 0.5f) * (ox * ox) + (curv_y * 0.5f) * (oy * oy) - oz;

    float t = 0.0f;
    const float eps = 1e-12f;
    bool valid = false;

    if (fabsf(A) < eps) {
        // Degenerate (linear) case.
        t = -C / B;
        valid = (t > 0.0f);
    }
    else {
        float discr = B * B - 4.0f * A * C;
        if (discr >= 0.0f) {
            float sqrt_discr = sqrtf(discr);
            float t1 = (-B - sqrt_discr) / (2.0f * A);
            float t2 = (-B + sqrt_discr) / (2.0f * A);
            // Choose the smallest positive t.
            if (t1 > 0.0f && t1 < t2) {
                t = t1;
                valid = true;
            }
            else if (t2 > 0.0f) {
                t = t2;
                valid = true;
            }
        }
    }

    // Discard if no valid t or if t is not within the ray's bounds.
    if (!valid || t < ray_tmin || t > ray_tmax) {
        return;
    }

    //
    // Compute the local intersection coordinates.
    //
    float x_hit = ox + t * dx;
    float y_hit = oy + t * dy;
    // (Optionally, you could compute z_hit = oz + t*dz and verify it is near f(x,y).)

    //
    // Check if the hit is within the rectangle's flat bounds.
    // The parametric coordinates are:
    //    a1 = x_hit / (L1/2)   and   a2 = y_hit / (L2/2)
    //
    float a1 = x_hit / (L1/2.);
    float a2 = y_hit / (L2/2.);
    if (a1 < -1.0f || a1 > 1.0f || a2 < -1.0f || a2 > 1.0f) {
        return;
    }

    //
    // Compute the surface normal at the hit on the paraboloid.
    // The height function is:
    //    f(x,y) = (curv_x/2)*x^2 + (curv_y/2)*y^2
    // so its partial derivatives are:
    //    f_x = curv_x * x    and    f_y = curv_y * y.
    // Then the (unnormalized) local normal is:
    //    N_local = (-f_x, -f_y, 1) = ( -curv_x*x_hit, -curv_y*y_hit, 1 ).
    //
    float3 N_local = normalize(make_float3(-curv_x * x_hit,
        -curv_y * y_hit,
        1.0f));
    // Transform the normal back to world coordinates.
    float3 world_normal = normalize(N_local.x * e1 +
        N_local.y * e2 +
        N_local.z * n);

    // Compute the hit point in world space.
    float3 world_hit = ray_orig + t * ray_dir;

    // Report the intersection.
    // Here, the two reported extra attributes are the parametric coordinates (a1, a2),
    // encoded as unsigned integers.
    optixReportIntersection(t, 0,
        __float_as_uint(world_normal.x),
        __float_as_uint(world_normal.y),
        __float_as_uint(world_normal.z));    
}


// intersection algorithm for a flat triangle based on "Fast, Minimum Storage Ray/Triangle Intersection" by M�ller and Trumbore (1997)
// code from here: https://en.wikipedia.org/wiki/M%C3%B6ller%E2%80%93Trumbore_intersection_algorithm 
extern "C" __global__ void __intersection__triangle_flat()
{
	const OptixCSP::GeometryDataST::Triangle_Flat& tri = params.geometry_data_array[optixGetPrimitiveIndex()].getTriangle_Flat();

    const float3 ro = optixGetObjectRayOrigin();
    const float3 rd = optixGetObjectRayDirection();

	//printf("Ray origin: (%f,%f,%f), direction: (%f,%f,%f)\n", ro.x, ro.y, ro.z, rd.x, rd.y, rd.z);

    const float3 edge1 = tri.e1;
    const float3 edge2 = tri.e2;


    const float3 pvec = cross(rd, edge2);
    const float  det = dot(edge1, pvec);

    // Backface culling + parallel rejection
    // (det must be strictly positive and not tiny)
    const float eps = 1e-8f;
    if (det <= eps) return;

    const float inv_det = 1.0f / det;

    const float3 tvec = ro - tri.v0;
    const float  u = dot(tvec, pvec) * inv_det;
    if (u < 0.0f || u > 1.0f) return;

    const float3 qvec = cross(tvec, edge1);
    const float  v = dot(rd, qvec) * inv_det;
    if (v < 0.0f || (u + v) > 1.0f) 
        return;

    const float  t = dot(edge2, qvec) * inv_det;
    if (t < optixGetRayTmin() || t > optixGetRayTmax()) return;

    float3 world_normal = tri.normal;

    optixReportIntersection(t, 0,
        __float_as_uint(world_normal.x),
        __float_as_uint(world_normal.y),
        __float_as_uint(world_normal.z));

}
